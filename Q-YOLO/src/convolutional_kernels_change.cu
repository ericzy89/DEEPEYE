#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

//
__global__ void quantize_kernel(float *x, int n, float *quantize,int kbits)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    static int integer1,integer2;
    integer1 = 1 << kbits;

    static float temp,temp1,temp2;
    temp=x[i];
    if(temp>1)         quantize[i] = 1;
    else if(temp<-1)   quantize[i] = -1;
    else
    {
       temp1 = (temp + 1) / 2.0;
       temp2 = temp1 * (integer1 - 1);
       integer2 = temp2 + 0.5;
       quantize[i] = (2.0 / (integer1 - 1) * integer2 - 1);
    }
}

void quantize_gpu(float *x, int n, float *quantize,int kbits)
{
    quantize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, quantize,kbits);
    check_error(hipPeekAtLastError());
}

__global__ void quantize_kernel_second(float *x, int n, float *quantize,int kbits)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    static int integer1,integer2;
    integer1 = 1 << kbits;

    static float temp,temp1,temp2;
    temp = x[i];
    if(temp>1)         quantize[i] = 1;
    else if(temp<-1)   quantize[i] = -1;
    else
    {
       temp1 = temp * integer1;
       integer2 = temp1;
       quantize[i] = (float) integer2 / (float) integer1;
    }
}

void quantize_gpu_second_method(float *x, int n, float *quantize,int kbits)
{
    quantize_kernel_second<<<cuda_gridsize(n), BLOCK>>>(x, n, quantize,kbits);
    check_error(hipPeekAtLastError());
}
//

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

//
__global__ void three_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    int i = 0;
    float mean = 0;
    int count=0;
    float sum=0;

    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean =mean / size;
    mean *= 0.7;


    for(i = 0; i < size; ++i){
        if( fabsf( weights[f*size + i] ) >= mean)
        {
            sum += fabsf(weights[f*size + i]);
            count++;
        }
    }

    sum = sum / count;


    for(i = 0; i < size; ++i){
       if(weights[f*size + i] >= mean)
          binary[f*size + i]=sum;
       else if(weights[f*size + i] <= -mean)
          binary[f*size + i]=-sum;
       else
          binary[f*size + i]=0;
    }

}

void three_weights_gpu(float *weights, int n, int size, float *binary)
{
    three_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}
//

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabsf(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}



void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    int lgy,temp,lll;
    int number_of_weights = l.nweights;
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    //
    if(l.three){
        three_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.three_weights_gpu);
        swap_three(&l);
    }
    //

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
    }

    //
    if(l.quantize)
    {
       three_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.three_weights_gpu);
       quantize_gpu_second_method(net.input_gpu, l.c*l.h*l.w*l.batch, l.quantize_input_gpu,7);
       swap_three(&l);
       net.input_gpu = l.quantize_input_gpu;
    }
    //


    /*
    quantize_gpu_second_method(net.input_gpu, l.c*l.h*l.w*l.batch, net.input_gpu,3);
    cuda_pull_array(net.input_gpu, net.input, l.inputs);
    for(lgy=0;lgy<100;lgy++)
    {
       printf("%10f",net.input[lgy]);
       if((lgy+1)%10==0) printf("\n");
    }
    printf("Input an integer number:");
    scanf("%d",&lll);
    */


    //
    if(l.print_weights){
        //three_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.weights_gpu);
        pull_convolutional_layer(l);
        printf("Layer total weights number %d\n",number_of_weights);
        for(lgy=0;lgy<number_of_weights;lgy++)
        {
           printf("%10f",l.weights[lgy]);
           if((lgy+1)%10==0) printf("\n");
        }
        printf("\n\n");
    }
    //

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.weights_gpu + j*l.nweights/l.groups;
            float *b = net.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;

            im2col_gpu(net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w,
                l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //
    /*if(l.print_activations){
      cuda_pull_array(l.output_gpu, l.output, l.outputs);
      FILE *fp;
      fp=fopen("~/test/"+video_name+"_"+frame_index+".txt","w");
      fp=fopen("a.txt","w");
      frame_index++;
      for (lgy = 0; lgy < l.outputs; lgy++) {  //将数组中的整数写入fp指向的txt文件
        fprintf(fp,"%10f\n",l.output[lgy]);
      }
      fclose(fp);*/
       // printf("Layer outputs total number:%d\n",l.outputs);
       // scanf("%d",&temp);
       // cuda_pull_array(l.output_gpu, l.output, l.outputs);
       // for(lgy=0;lgy<l.outputs;lgy++)
       // {
       //     printf("%10f",l.output[lgy]);
       //     if((lgy+1)%10==0) printf("\n");
       // }
       // printf("\n\n");
    }
    //

    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
    //
    if(l.three || l.quantize) swap_three(&l);
    //
}

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size/2.f);
    int h_offset = -(size/2.f);

    int out_index = j + w*(i + h*(k + c*b));
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w*(cur_h + h*(k + b*c));
            int valid = (cur_h >= 0 && cur_h < h &&
                    cur_w >= 0 && cur_w < w);
            delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
        }
    }
}

extern "C" void smooth_layer(layer l, int size, float rate)
{
    int h = l.out_h;
    int w = l.out_w;
    int c = l.out_c;

    size_t n = h*w*c*l.batch;

    smooth_kernel<<<cuda_gridsize(n), BLOCK>>>(l.output_gpu, n, l.w, l.h, l.c, size, rate, l.delta_gpu);
    check_error(hipPeekAtLastError());
}

void backward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
    //
    if(l.quantize) net.input_gpu = l.quantize_input_gpu;
    //
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            net.input_gpu,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        //
        if(l.three || l.quantize) swap_three(&l);
        //
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        //
        if(l.three || l.quantize) swap_three(&l);
        //
        if(l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
        //
        if(l.quantize) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
        //
    }

#else
    int m = l.n/l.groups;
    int n = l.size*l.size*l.c/l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.delta_gpu + (i*l.groups + j)*m*k;
            float *b = net.workspace;
            float *c = l.weight_updates_gpu + j*l.nweights/l.groups;

            float *im = net.input_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;

            im2col_gpu(im, l.c/l.groups, l.h, l.w,
                    l.size, l.stride, l.pad, b);
            gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

            if(net.delta_gpu){
                if(l.binary || l.xnor) swap_binary(&l);
                //
                if(l.three || l.quantize) swap_three(&l);
                //
                a = l.weights_gpu + j*l.nweights/l.groups;
                b = l.delta_gpu + (i*l.groups + j)*m*k;
                c = net.workspace;

                gemm_gpu(1,0,n,k,m,1,a,n,b,k,0,c,k);

                col2im_gpu(net.workspace, l.c/l.groups, l.h, l.w, l.size, l.stride,
                    l.pad, net.delta_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w);
                if(l.binary || l.xnor) {
                    swap_binary(&l);
                }
                //
                if(l.three  ||  l.quantize) swap_three(&l);
                //
            }
            if(l.xnor) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
            //
            if(l.quantize) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
            //
        }
    }
#endif
}

void pull_convolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_convolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
}
